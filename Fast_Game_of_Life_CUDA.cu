#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <string.h>
#include <unistd.h>

#define KERNEL_LOCAL_SZ 8

#define MAXN 2048
char host_Mtx[2][MAXN*MAXN];

__device__ inline void move_border(char g[][8+2], char *IN,
    int localX, int localY, int localSz, int x, int y, int binN) {
    if (localX == 1) {
        g[localX-1][localY] = IN[(x-1) * binN + y];
        if (localY == 1)
            g[localX-1][localY-1] = IN[(x-1) * binN + (y-1)];
        if (localY == localSz)
            g[localX-1][localY+1] = IN[(x-1) * binN + (y+1)];
    }
    if (localY == 1)    g[localX][localY-1] = IN[x * binN + (y-1)];
    if (localY == localSz)    g[localX][localY+1] = IN[x * binN + (y+1)];
    if (localX == localSz) {
        g[localX+1][localY] = IN[(x+1) * binN + y];
        if (localY == 1)
            g[localX+1][localY-1] = IN[(x+1) * binN + (y-1)];
        if (localY == localSz)
            g[localX+1][localY+1] = IN[(x+1) * binN + (y+1)];
    }
}
__global__ void simulate(char *IN, char *OUT, int *binN,int *N) {
    int localX = threadIdx.x+1;
    int localY = threadIdx.y+1;
    int x = blockIdx.x*8+localX;
    int y = blockIdx.y*8+localY;
    int localSz = 8;//8
    // printf("x = %d,y = %d, localX = %d, locaY = %d, localSz = %d\n",x,y,localX,localY,localSz);
    __shared__ char g[8+2][8+2];
 
    const int dx[] = {-1, -1, -1, 0, 0, 1, 1, 1};
    const int dy[] = {-1, 0, 1, -1, 1, -1, 0, 1};
 
    // move itself to local
    char t = IN[x * (*binN) + y];
    g[localX][localY] = t;
    // move border to local
    move_border(g, IN, localX, localY, localSz, x, y,*binN);
    __syncthreads();
    // barrier(CLK_LOCAL_MEM_FENCE);
 
    if (x > *N || y > *N)    return ;
 
    int adj = 0;
    for (int i = 0; i < 8; i++) {
        int cx = localX + dx[i];
        int cy = localY + dy[i];
        adj += g[cx][cy];
    }
    OUT[x * (*binN) + y] = (t == 0 && adj == 3) || (t == 1 && (adj == 2 || adj == 3));
}

int main(void)
{
  char *device_IN[2];
  int * device_binN;
  int * device_N;
  int binN;
  int size = sizeof(char) * MAXN*MAXN;

  hipMalloc((void **)&device_IN[0], size);
  hipMalloc((void **)&device_IN[1], size);
  hipMalloc((void **)&device_binN,(int)sizeof(int));
  hipMalloc((void **)&device_N,(int)sizeof(int));

  int N,M;
  assert(scanf("%d %d", &N, &M) == 2);
  static char str[2048][2048];
  for (int i = 1; i <= N; i++)
      assert(scanf("%s", str[i]+1) == 1);
  binN = N+2;
  while (binN % KERNEL_LOCAL_SZ)
      binN++;
  // printf("binN = %d\n",binN );
  memset(host_Mtx, 0, sizeof(host_Mtx));
  for (int i = 1; i <= N; i++) {
      for (int j = 1; j <= N; j++)
          host_Mtx[0][i*binN + j] = str[i][j] - '0';
  } 
  // for (int i = 1; i <= N; i++) {
  //       for (int j = 1; j <= N; j++)
  //           putchar(host_Mtx[0][i*binN+j]+'0');
  //       puts("");
  //   }

  int flag = 0;
  dim3 block(KERNEL_LOCAL_SZ, KERNEL_LOCAL_SZ);
  dim3 grid(binN/KERNEL_LOCAL_SZ, binN/KERNEL_LOCAL_SZ);
  hipMemcpy(device_IN[0], host_Mtx[0], size, hipMemcpyHostToDevice);
  hipMemcpy(device_IN[1], host_Mtx[1], size, hipMemcpyHostToDevice);
  hipMemcpy(device_binN, &binN, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_N, &N, sizeof(int), hipMemcpyHostToDevice);
  for (int it = 0; it < M; it++) {
        // -- set argument to kernel
        // clStat = clSetKernelArg(clKrn[0], 0, sizeof(cl_mem), &clMemIn[0][flag]);
        // CheckFailAndExit(clStat);
        // clStat = clSetKernelArg(clKrn[0], 1, sizeof(cl_mem), &clMemIn[0][!flag]);
        // CheckFailAndExit(clStat);
 
  simulate <<< grid, block >>> ((char *)device_IN[flag], (char *)device_IN[!flag],(int *)device_binN,(int*)device_N);
        // -- execute
        // clStat = clEnqueueNDRangeKernel(clQue[0], clKrn[0], 2, globalOffset,
        //         globalSize, localSize, 0, NULL, NULL);
        // CheckFailAndExit(clStat);
        flag = !flag;
  }
  hipMemcpy(host_Mtx[flag], device_IN[flag], sizeof(char)*binN*binN, hipMemcpyDeviceToHost);
  // cudaMemcpy(host_Mtx[flag], device_IN[flag], size, cudaMemcpyDeviceToHost);
    // -- read back
    // clStat = clEnqueueReadBuffer(clQue[0], clMemIn[0][flag], CL_TRUE, 0,
    //         sizeof(char)*binN*binN, hostMtx[flag], 0, NULL, NULL);
    for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= N; j++)
            putchar(host_Mtx[flag][i*binN+j]+'0');
        puts("");
    }
        // puts("");


  


  hipFree(device_IN[0]);
  hipFree(device_IN[1]);
  hipFree(device_binN);
  hipFree(device_N);
  // free(host_A);
  // free(host_B);
  // free(host_C);
}
			
