#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <inttypes.h>
#include <stdint.h>
__device__ static inline uint32_t rotate_left(uint32_t x, uint32_t n) {
    return  (x << n) | (x >> (32-n));
}
__device__ static inline uint32_t encrypt(uint32_t m, uint32_t key) {
    return (rotate_left(m, key&31) + key)^key;
}
 
#define MAXN 16777216
#define Size 512
#define Block MAXN/Size


__global__ void hello(uint32_t C[Block],int* N,uint32_t* key1,uint32_t* key2)
{
  uint32_t sum=0 ,a,b;
  int left = blockIdx.x*Size;
  int right = (blockIdx.x+1)*Size < *N ? (blockIdx.x+1)*Size : *N;
  for (int k = left; k < right; k++){
    a = encrypt(k, *key1);
    b = encrypt(k, *key2);
    sum += a * b;
  }
  C[blockIdx.x] = sum;
}

int main(void)
{
  int *device_N;
  uint32_t *device_C,*device_key1,*device_key2;
  uint32_t  *host_C,*host_key1,*host_key2;
  int *host_N;
  int size = sizeof(uint32_t) * Block;

  // dim3 blocks(Size, Size);

  // cudaMalloc((void *)&device_A, size);
  // cudaMalloc((void *)&device_B, size);
  hipMalloc((void **)&device_C, size);
  hipMalloc((void **)&device_N, (int)sizeof(int));
  hipMalloc((void **)&device_key1, (int)sizeof(uint32_t));
  hipMalloc((void **)&device_key2, (int)sizeof(uint32_t));

  // host_A = (uint32_t *)malloc(size);
  // host_B = (uint32_t *)malloc(size);
  host_C = (uint32_t *)malloc(size);
  host_N = (int *)malloc(sizeof(int));
  host_key1 = (uint32_t *)malloc(sizeof(uint32_t));
  host_key2 = (uint32_t *)malloc(sizeof(uint32_t));


    while (scanf("%d %" PRIu32 " %" PRIu32, host_N, host_key1, host_key2) == 3) {
        // int chunk = N / 4;
        // for (int i = 0; i < N; i++) {
        //     A[i] = encrypt(i, key1);
        //     B[i] = encrypt(i, key2);
        // }
 
        // for (int i = 0; i < N; i++)
        //     C[i] = A[i] * B[i];
       hipMemcpy(device_N,host_N,sizeof(int),hipMemcpyHostToDevice);
      hipMemcpy(device_key1,host_key1,sizeof(uint32_t),hipMemcpyHostToDevice);
      hipMemcpy(device_key2,host_key2,sizeof(uint32_t),hipMemcpyHostToDevice);
        int ex_N = (*host_N-1) / Size + 1;
          hello <<< ex_N, 1 >>> ((uint32_t *) device_C,(int *)device_N,(uint32_t*)device_key1,(uint32_t *)device_key2);
      hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);


      uint32_t sum = 0;
        for (int i = 0; i < ex_N; i++)
            sum += host_C[i];
        printf("%" PRIu32 "\n", sum);
    }

  // cudaMemcpy(device_A, host_A, size, cudaMemcpyHostToDevice);
  // cudaMemcpy(device_B, host_B, size, cudaMemcpyHostToDevice);



  // int k = 0;
  // for (int i = 0; i < Size; i++)
  //   for (int j = 0; j < Size; j++)
  //     printf("host_C[%d][%d] = %d\n", i, j, host_C[k++]);

  // cudaFree(device_A);
  // cudaFree(device_B);
  hipFree(device_C);
  hipFree(device_N);
  hipFree(device_key1);
  hipFree(device_key2);
  // free(host_A);
  // free(host_B);
  free(host_C);
  free(host_N);
  free(host_key1);
  free(host_key2);
}
      
